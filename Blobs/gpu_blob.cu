#include "hip/hip_runtime.h"
/*
	(15,16) ---> 15*16 = 240 
	(16,20) ---> 16*20 = 320
	
	each block will be of dimension 15 x 16.

	(20,20)

	In the shared memory the labelSharedMemory contains the index value of the pixel as root. Labelling is done by storing value of indices at the pixels.
	The index value stored is the one that is minimum from its neighbouring 8 pixels.
							
	-------------------------------------------------------------------------
	|							Shared Memory								|
	|	-------------------------		-----------------------------		|
	|	|						|		|							|		|
	|	|						|		|							|		|
	|	|		Label			|		|		  Segment 			|		|
	|	|	Shared Memory		|		|	   Shared Memory    	|		|
	|	|						|		|							|		|
	|	|						|		|							|		|
	|	|						|		|							|		|
	|	-------------------------		-----------------------------		|
	-------------------------------------------------------------------------

	The best way to do labelling is using disjoint set datasctructure(Union Find DS).
	See Wikipidea
*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
#include "../API/api.h"

inline __device__ int findRoot(int* buf, int x) 
{
	int nextX;
    do {
	  nextX = x;
      x = buf[nextX];
    } while (x < nextX);
    return x;    
}

__device__ bool flag = true;
inline __device__ void gpuMin( int *temp, int newLabel)
{
	while( flag !=true){}
	flag = false;
	*temp = min( *temp, newLabel);
	flag = true;
}

__device__ bool flag1 = true;
inline __device__ void gpuMin1( int *temp, int newLabel)
{
	while( flag1 !=true){}
	flag1 = false;
	*temp = min( *temp, newLabel);
	flag1 = true;
}

inline __device__ void unionF(int* buf, unsigned char *buf_uchar, unsigned char seg1, unsigned char seg2, int reg1, int reg2, int* changed)
{
	if(seg1 == seg2) 
	{			
		int newReg1 = findRoot(buf, reg1);		
		int newReg2 = findRoot(buf, reg2);	
	
		if(newReg1 > newReg2) {			
			atomicMin(buf+newReg1, newReg2);		
			//gpuMin(buf+newReg1,newReg2);
			buf_uchar[newReg1] = min( buf_uchar[newReg1], newReg2);		
			changed[0] = 1;			
		} else if(newReg2 > newReg1) {		
			atomicMin(buf+newReg2, newReg1);	
			//gpuMin1(buf+newReg2,newReg2);
			buf_uchar[newReg2] = min( buf_uchar[newReg2], newReg2);
			changed[0] = 1;
		}			
	} 	
}

/*
__shared__ __device__ bool atmFlag = false;
inline __device__ int gpuMin( int *temp, int newLabel)
{
	*temp = min( *temp, newLabel);

}
*/

texture<unsigned char, 2, hipReadModeElementType> texSrc;


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/*													  Local labelling of Blobs 															   */
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void cclSharedLabelling( unsigned char *gpu_in, int *gpu_labels, unsigned char *gpu_labels_uchar, const int pitch, const int segOff, const int dataWidth)
{
	int 	  x = blockIdx.x * blockDim.x + threadIdx.x;
    int       y = blockIdx.y * blockDim.y + threadIdx.y;
    int  offset = x + y * blockDim.x * gridDim.x;
    int shPitch = blockDim.x + 2;	// This is the pitch for shared memory of labels.

    int    localIndex = threadIdx.x + 1 + (threadIdx.y + 1) * shPitch; // This is the local address inside shared memory that has 1 pixel width of apron.
    int      newLabel = localIndex;
    int      oldLabel = 0;
    int         index = x + y * pitch; // This is the address of the pixel in global memory
    int segLocalIndex = localIndex + segOff;

	// This is the new syntax for declaring shared memory //
    extern __shared__ int sMem[];

    //shared flag that is used to check for the final solution on the processed tile 
	//if there are any two connected elements with different labels the flag is set to 1
	__shared__ int sChanged[1];

	////// Initializing the shared memory. Setting the boundary values to 0 i.e background /////
	if(threadIdx.x == blockDim.x-1) 
	{	
		sMem[localIndex+1] = 0;
		sMem[segLocalIndex+1] = 0;
	}
	if(threadIdx.x == 0) 
	{	
		sMem[localIndex-1] = 0;
		sMem[segLocalIndex-1] = 0;
	}
	if(threadIdx.y == blockDim.y-1) {			
		sMem[localIndex+shPitch] = 0;
		sMem[segLocalIndex+shPitch] = 0;

		if(threadIdx.x == 0) {			
			sMem[localIndex+shPitch-1] = 0;
			sMem[segLocalIndex+shPitch-1] = 0;
		}
		if(threadIdx.x == blockDim.x-1) {			
			sMem[localIndex+shPitch+1] = 0;
			sMem[segLocalIndex+shPitch+1] = 0;
		}	
	}
	if(threadIdx.y == 0) {			
		sMem[localIndex-shPitch] = 0;
		sMem[segLocalIndex-shPitch] = 0;
		if(threadIdx.x == 0) {			
			sMem[localIndex-shPitch-1] = 0;
			sMem[segLocalIndex-shPitch-1] = 0;
		}
		if(threadIdx.x == blockDim.x-1) {			
			sMem[localIndex-shPitch+1] = 0;
			sMem[segLocalIndex-shPitch+1] = 0;
		}	
	}

	/// VVV IMP: I think that the variables declared inside a __global__ function call are register variables, and not normal variables.
	/// The register variables are faster than shared memory. But dont overuse it due to threads throughput.
	unsigned char pixel;
	unsigned char nPixel[8];	// The neighbouring pixels.

	// Current pixel retrieved for processing //
	//pixel = tex2D( texSrc, x, y);  // This is with usage of textures.
	pixel = gpu_in[ offset ];	// This is with global memory call.
	
	sMem[segLocalIndex] = (int)pixel;// This step will load the segmentation shared memory with all the required pixels
	__syncthreads();

	//store data about segments into registers so that we don't have to access shared memory
	//(the data are never modified)
	nPixel[0] = sMem[segLocalIndex-shPitch-1];
	nPixel[1] = sMem[segLocalIndex-shPitch];
	nPixel[2] = sMem[segLocalIndex-shPitch+1];
	nPixel[3] = sMem[segLocalIndex-1];
	nPixel[4] = sMem[segLocalIndex+1];
	nPixel[5] = sMem[segLocalIndex+shPitch-1];
	nPixel[6] = sMem[segLocalIndex+shPitch];
	nPixel[7] = sMem[segLocalIndex+shPitch+1];

	while(!NULL)
	{
		//in first pass the newLabel is equal to the local address of the element
		sMem[localIndex] = newLabel;

		//reset the check flag for each block
		if((threadIdx.x | threadIdx.y) == 0) sChanged[0] = 0;
		oldLabel = newLabel;
		__syncthreads();

		//if the element is not a background, compare the element's label with its neighbors
		if(pixel != 0) 
		{	
			if( pixel == nPixel[0])
				newLabel = min( newLabel, sMem[localIndex-shPitch-1]);
			if( pixel == nPixel[1])
				newLabel = min( newLabel, sMem[localIndex-shPitch]);
			if( pixel == nPixel[2])
				newLabel = min( newLabel, sMem[localIndex-shPitch+1]);
			if( pixel == nPixel[3])
				newLabel = min( newLabel, sMem[localIndex-1]);
			if( pixel == nPixel[4])
				newLabel = min( newLabel, sMem[localIndex+1]);
			if( pixel == nPixel[5])
				newLabel = min( newLabel, sMem[localIndex+shPitch-1]);
			if( pixel == nPixel[6])
				newLabel = min( newLabel, sMem[localIndex+shPitch]);
			if( pixel == nPixel[7])
				newLabel = min( newLabel, sMem[localIndex+shPitch+1]);
				
		}
		__syncthreads();

		if( oldLabel > newLabel)
		{
			//if there is a neigboring element with a smaller label, update the equivalence tree of the processed element
			//(the tree is always flattened in this stage so there is no need to use findRoot to find the root)	
			//VVVVIMP: This step is like merging of two trees together.				
            //Be carefull when removing this function. Atomic is used to prevent multiple threads from accessing same memory.
			//It is like a particualar thread has acquired a lock on the address.			
			atomicMin(sMem+oldLabel, newLabel); 
			//sMem[localIndex] = newLabel;
			//sMem[oldLabel] = min( sMem[oldLabel], newLabel);
			
			//set the flag to 1 bcoz it is necessary to perform another iteration of the CCL solver
			sChanged[0] = 1;
		}
		__syncthreads();

		if(sChanged[0] == 0) break;

		//flatten the equivalence tree
		newLabel = findRoot(sMem,newLabel);			
		__syncthreads();
	
	}	

	if(pixel == 0) newLabel = -1;	 // This is the labelling of the background pixel.
	else
	{	// The following loop translates each local label to a unique global label.
		//transfer the label into global coordinates 
		y = newLabel / (blockDim.x+2);
		x = newLabel - y*(blockDim.x+2);
		x = blockIdx.x*blockDim.x + x-1;
		y = blockIdx.y*blockDim.y + y-1;
		newLabel = x+y*dataWidth;	
	}	

	gpu_labels[index] = newLabel;
	gpu_labels_uchar[index] = (unsigned char)newLabel;

}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/*													Flattening of all the elements															*/
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void flattenEquivalenceTreesKernel(int* gpu_labels_out, int* gpu_labels_in, unsigned char *gpu_labels_uchar, uint pitch, const int dataWidth)												
{
	uint     x = (blockIdx.x*blockDim.x)+threadIdx.x;
    uint     y = (blockIdx.y*blockDim.y)+threadIdx.y;  
    uint index = x+y*pitch;
    uint label = gpu_labels_in[index];

	uint newLabel;

	if((label != -1) && (label != index))
	{
		newLabel = findRoot( gpu_labels_in, label);

		if(newLabel < label)
		{
			gpu_labels_out[index] = newLabel;
			gpu_labels_uchar[index] = (unsigned char)newLabel;
		}
	}
}

void flattenTrees( int *gpu_labels, unsigned char *gpu_labels_uchar, int threadsX, int threadsY, int imageW, int imageH)
{	
	dim3 block(threadsX, threadsY, 1);
    dim3 grid(imageW / block.x, imageH / block.y, 1);

    flattenEquivalenceTreesKernel<<<grid, block>>>( gpu_labels, gpu_labels, gpu_labels_uchar, imageW, imageW);
}

/////////////////////////////////////////////// Merge Borders ////////////////////////////////////////////////////////////////////////

__global__ void merge( int *gpu_labels, unsigned char *gpu_labels_uchar, int tileDim, const int pitch)
{

	int xT = (blockIdx.x * blockDim.x) + threadIdx.x;
	int yT = (blockIdx.y * blockDim.y) + threadIdx.y;

	__shared__ int sChanged[1];


	// horizontal bottom border

	uint 	  x = (xT) * tileDim + threadIdx.z;
	int offset = (threadIdx.x * tileDim) + threadIdx.z;
	uint 	  y = ((yT+1) * tileDim)-1;
	int 	idx = x + y * pitch;
		
	unsigned char seg = tex2D(texSrc, x, y);

	while(!NULL)
	{

		if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		{
			sChanged[0] = 0;			
		}
		__syncthreads();

		
		if( seg != 0)
		{
			if(offset > 0) unionF( gpu_labels, gpu_labels_uchar, seg, tex2D(texSrc,x-1,y+1), idx, idx+pitch-1, sChanged);
			unionF( gpu_labels, gpu_labels_uchar, seg, tex2D(texSrc,x,y+1), idx, idx+pitch, sChanged);
			if(offset < (blockDim.x*tileDim)) unionF( gpu_labels, gpu_labels_uchar, seg, tex2D(texSrc,x+1,y+1), idx, idx+pitch+1, sChanged);
		}

		// vertical right border	

			 x = ((xT+1)*tileDim)-1;
			 y = (yT*tileDim) + threadIdx.z;
		offset = (threadIdx.y * tileDim)+threadIdx.z;
		   idx = x + y * pitch;
		
		seg = tex2D(texSrc, x, y);

		if( seg != 0)
		{
		if( offset > 0 ) unionF( gpu_labels, gpu_labels_uchar, seg, tex2D(texSrc,x+1,y-1), idx, idx-pitch+1, sChanged);
			unionF( gpu_labels, gpu_labels_uchar, seg, tex2D(texSrc,x+1,y), idx, idx+1, sChanged);
			if(offset < (blockDim.y*tileDim)) unionF( gpu_labels, gpu_labels_uchar, seg, tex2D(texSrc,x+1,y+1), idx, idx+pitch+1, sChanged);
		}

		__syncthreads();
		
		if(sChanged[0] == 0) 		
			break;	
		
		__syncthreads();
	}
}

void mergeBorders( int *gpu_labels, unsigned char *gpu_labels_uchar, int threadsX, int threadsY, int imageW, int imageH)
{

	int xTiles = 4;
	int yTiles = 4;
	int threadsPerBlock = threadsX;	// This denotes the no. of pixels in borders to be merged at a time.... If the size of border is large we can also increment these threads
	int tileSize = threadsX;

	dim3 block(xTiles,yTiles,threadsPerBlock);
	dim3 grid(imageW/(block.x*block.z), imageH/(block.y*block.z));

	merge<<<grid,block>>>( gpu_labels, gpu_labels_uchar, tileSize, imageW);	// FIXME: Try changing this value to 240*sizeof(int)
	
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/*                 									Main Wrapper about the function   								  				  */
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
gpu_error_t gpu_DetectBlob( gpu_context_t *ctx)
{

	gpu_error_t err = GPU_OK;

	int   imageW = ctx->width;
	int   imageH = ctx->height;
	int threadsX = 20;
	int threadsY = 20;
	int    temp1 = imageW/4;
	int    temp2 = imageH/4;	
	int 	   i = 15;

	if( (imageW==480 || imageW==240) && (imageH==320 || imageH==640))
	{
		threadsX = 20;
		threadsY = 20;
	}
	else
	{
		while( (temp1%i != 0) || (temp2%i != 0))
		{
			i++;
			if(i>20)
				break;
		}
		if( i>20 )
		{
			fprintf(stderr,"Invalid dimensions for blob detection");
			exit(EXIT_FAILURE);
		}
		threadsX = i;
		threadsY = i;				
	}	

	
//	float elapsedtime;
//	hipEvent_t start, stop;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
	
    int *gpu_labels;
    hipMalloc( (void **)&gpu_labels, imageW * imageH * sizeof(int));

	err = checkCudaError();
	if( err != GPU_OK)
		return err;

    int *labels_int;
    labels_int = (int *)malloc(imageW*imageH*sizeof(int));

    //// This buffer is temporary and only used for debuggin purpose ////
    unsigned char *gpu_labels_uchar;
    hipMalloc( (void **)&gpu_labels_uchar, imageW * imageH * sizeof(unsigned char));

   	err = checkCudaError();
	if( err != GPU_OK)
		return err;
	//////
   
//    hipEventRecord(start,0);

    //////////////////////////////////////////// Local Shared Labelling /////////////////////////////////////////////////////
    dim3 threads(threadsX,threadsY);
    dim3 blocks( imageW/threadsX, imageH/threadsY);

    int labelSize = (threads.x + 2) * (threads.y + 2) * sizeof(int); //This is the size for storage of labels to the corresponding pixels
    int   segSize = (threads.x + 2) * (threads.y + 2) * sizeof(int); //This is the size of storage for segments.
    
	cclSharedLabelling<<< blocks, threads, (labelSize + segSize)>>>( ctx->gpu_buffer_1, gpu_labels, gpu_labels_uchar, 240, labelSize/sizeof(int), 240);

	err = checkCudaError();
	if( err != GPU_OK)
		return err;
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	
	//////////////////////////////////////////// Merging Blobs Together /////////////////////////////////////////////////////
	
	hipArray *src;
    hipChannelFormatDesc floatTex = hipCreateChannelDesc<unsigned char>();
    hipMallocArray(&src, &floatTex, imageW, imageH);
    hipMemcpyToArray(src, 0, 0, ctx->gpu_buffer_1, imageW * imageH, hipMemcpyDeviceToDevice);
    hipBindTextureToArray(texSrc, src);  

	mergeBorders( gpu_labels, gpu_labels_uchar, threadsX, threadsY, imageW, imageH);

	err = checkCudaError();
	if( err != GPU_OK)
		return err;
 
    hipUnbindTexture(texSrc);
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


	//////////////////////////////////////// Updating all the labels (i.e flattening )/////////////////////////////////////////////////
	flattenTrees( gpu_labels, gpu_labels_uchar, threadsX, threadsY, imageW, imageH);
	err = checkCudaError();
	if( err != GPU_OK)
		return err;
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//	hipEventRecord(stop,0);
//	hipEventSynchronize(stop);
//	hipEventElapsedTime(&elapsedtime,start,stop);
//	hipEventDestroy(start);
//	hipEventDestroy(stop);

	hipMemcpy( ctx->output_buffer_1, gpu_labels_uchar, imageW*imageH, hipMemcpyDeviceToHost);
	err = checkCudaError();
	if( err != GPU_OK)
		return err;
//	hipMemcpy( labels_int, gpu_labels, imageW*imageH*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(gpu_labels);
	hipFree(gpu_labels_uchar);

	return err;
}
