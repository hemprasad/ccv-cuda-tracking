#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../API/api.h"
#include "assert.h"
#include "stdio.h"

///////////////////////// Threshold Cuda function ////////////////////////
__global__ void convert( int width, int height, int th_value, unsigned char *gpu_in)
{
	int 	tx = threadIdx.x + __mul24(blockIdx.x, blockDim.x);
	int 	ty = threadIdx.y + __mul24(blockIdx.y, blockDim.y);
	int offset = tx + ty * __mul24(blockDim.x, gridDim.x);

	if( offset < width * height )
	{
		if( (gpu_in[offset]) < th_value )
			gpu_in[offset] = 0;
		else
			gpu_in[offset] = 255;
	}

}

/////////////////////////// Cuda Function Call wrapper ///////////////////////////////
gpu_error_t gpu_threshold( gpu_context_t *ctx, int th_value)
{
	assert(th_value);

	float elapsedtime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	gpu_error_t error = GPU_OK;
	int threadsX = ctx->threadsX;
	int threadsY = ctx->threadsY;
	int temp1 = ((ctx->width % threadsX) != 0 ? (ctx->width / threadsX) + 1 : ctx->width / threadsX );
	int temp2 = ((ctx->height % threadsY) != 0 ? (ctx->height / threadsY) + 1 : ctx->height / threadsY );

	///////////////////////////// Kernel Call ///////////////////////////////////////
	dim3 blocks(threadsX,threadsY);
	dim3 threads(temp1,temp2);	
	convert<<<blocks,threads>>>( ctx->width, ctx->height, th_value, ctx->gpu_buffer_1);
	/////////////////////////////////////////////////////////////////////////////////

	if( hipSuccess != hipMemcpy(ctx->output_buffer_1, ctx->gpu_buffer_1, ctx->width * ctx->height , hipMemcpyDeviceToHost))
		error = GPU_ERR_MEM;

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//FILE *file;
	//file = fopen("../timing.txt","a+");
	fprintf(stderr,"Threshold:%lf \n",elapsedtime);
	//fclose(file);

	return error;
}

