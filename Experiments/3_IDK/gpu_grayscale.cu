#include "hip/hip_runtime.h"

/////////////// Grayscale Cuda Fucntion ////////////////////
__global__ void convert(int width, int height, uchar4 *gpu_in)
{
	
	int tx = threadIdx.x + (blockIdx.x * blockDim.x);
	int ty = threadIdx.y + (blockIdx.y * blockDim.y);
	int offset = tx + ty * blockDim.x*gridDim.x;

	if(tx >= 240 || ty >= 320)
        return;	
		/*float color = 0.3 * (gpu_in[offset].x) + 0.6 * (gpu_in[offset].y) + 0.1 * (gpu_in[offset].z);
		gpu_in[offset].x = color;
		gpu_in[offset].y = color;
		gpu_in[offset].z = color;
		gpu_in[offset].w = 0;*/

	int offset4 = offset * 4;
		int c = *(unsigned int*)(&gpu_in[offset4]);
		int c1 = c / 16777216;
		int c2 = (c - (c1 * 16777216)) / 65536;
		int c3 = (c - (c1 * 16777216) - (c2 * 65536)) / 256;
		float color = 0.3f * c1 + 0.6f * c2 + 0.1f * c3;

		gpu_in[offset].x = (unsigned)color;
		gpu_in[offset].y = (unsigned)color;
		gpu_in[offset].z = (unsigned)color;
		gpu_in[offset].w = 0;
		
	
}
///////////////// CUDA function call wrapper /////////////////
float gpu_grayscale(int width, int height, unsigned char *in)
{
	uchar4 *gpu_in;
	float elapsedtime;
	hipMalloc((void **)&gpu_in, (width * height * 4 * sizeof(unsigned char)));
	hipMemcpy(gpu_in, in, (width * height * 4 * sizeof(unsigned char)), hipMemcpyHostToDevice);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	////////////////////////// Time consuming Task //////////////////////////////////	
	

	dim3 grid(18,18);
	dim3 block(16,16);
	convert<<<grid,block>>>(width, height, gpu_in);


	/////////////////////////////////////////////////////////////////////////////////

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
		hipMemcpy( in, gpu_in, (width * height * 4 * sizeof(unsigned char)), hipMemcpyDeviceToHost);
	hipFree(gpu_in);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return elapsedtime;
	
}

