#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "assert.h"

float elapsedTime;

static int flag = 0;
static unsigned char *staticBg = (unsigned char *)malloc(320*240);

texture<unsigned char, 2, hipReadModeElementType> texSrc;
texture<unsigned char, 2, hipReadModeElementType> texConstant;

/////////////////////////////////////////// Subtract Image ///////////////////////////////////////////////////////
__global__ void subtract( unsigned char *gpu_in, unsigned char *staticBg, unsigned char *outTemp)
{
	int  		ix = threadIdx.x + __mul24( blockIdx.x, blockDim.x);
	int  		iy = threadIdx.y + __mul24( blockIdx.y, blockDim.y);
	//float		 x = (float)ix + 0.5f; 
	//float 	 y = (float)iy + 0.5f;
	int 	offset = ix + iy * blockDim.x * gridDim.x;

	if(ix >= 240 || iy >= 320)
        return;
        
	//outTemp[offset] = tex2D(texSrc, x, y) - staticBg[offset];
//	outTemp[offset] = gpu_in[offset] - staticBg[offset];
	outTemp[offset] = ( (gpu_in[offset] - staticBg[offset]) < 0 ? 0 : (gpu_in[offset] - staticBg[offset]) );

}

void gpu_subtract( unsigned char *frameIn)
{
	//hipArray *src;
	//hipChannelFormatDesc tex = hipCreateChannelDesc<unsigned char>();
	//hipMallocArray(&src,&tex,240,320);
	//hipMemcpyToArray( src, 0, 0, in, 240*320, hipMemcpyHostToDevice);
	//hipBindTextureToArray(texSrc, src);

	
	unsigned char *in;
	in = (unsigned char *)malloc( 320 * 240 * sizeof(unsigned char));

	if(flag ==  0)
	{
	for( int i = 0; i < 320 * 240; i++)
	{
		staticBg[i] = frameIn[i];
	}
	}
	flag = 1;
	unsigned char *gpu_in;
	hipMalloc( (void **)&gpu_in, 240 * 320);
	hipMemcpy( gpu_in, frameIn, 240 * 320, hipMemcpyHostToDevice);
	
	unsigned char *temp;
	hipMalloc( (void **)&temp, 240 * 320);
	hipMemcpy( temp, staticBg, 240 * 320, hipMemcpyHostToDevice);

	unsigned char *outTemp;
	hipMalloc( (void **)&outTemp, 240 * 320);
	
    //////////////////////////////////////////////////////////////////////////////
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	dim3 threads( 16, 15);
	dim3 blocks( 15, 22);
	subtract<<< blocks, threads>>>( gpu_in, temp, outTemp);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	/////////////////////////////////////////////////////////////////////////////
	
	hipMemcpy( in, outTemp, 240 * 320, hipMemcpyDeviceToHost);
	for( int i = 0; i < 320 * 240; i++)
	{
		frameIn[i] = in[i];
	}
	//hipUnbindTexture(texSrc);

}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////

