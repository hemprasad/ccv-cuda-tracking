#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

float elapsedtime;
texture<uchar4, 2, hipReadModeElementType> texSrc;

////////////////////////////////////CUDA Programming///////////////////////////////////////////////////////////////

__global__ void convert(unsigned char *iin_1)
{

	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	int ty = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = tx + ty * blockDim.x * gridDim.x;
    const float  x = (float)tx + 0.5f;
    const float  y = (float)ty + 0.5f;

	if(tx >= 240 || ty >= 320)
        return;
	uchar4 temp;
	temp = tex2D(texSrc,x,y);
	float color = 0.3 * temp.x + 0.6 * temp.y + 0.1 * temp.z ;

	iin_1[offset] = color;
    
	/*
	int tx = threadIdx.x + (blockIdx.x * blockDim.x);
	int ty = threadIdx.y + (blockIdx.y * blockDim.y);
	int offset = tx + ty * blockDim.x*gridDim.x;

	if(offset < 240*320)
	{	
		float color = 0.3 * (gpu_in[offset].x) + 0.6 * (gpu_in[offset].y) + 0.1 * (gpu_in[offset].z);
		gpu_in[offset].x = color;
		gpu_in[offset].y = color;
		gpu_in[offset].z = color;
		gpu_in[offset].w = 0;
	}*/
	
	/*__shared__ unsigned char sh_Tile[16*16*4];
	
	int tx = threadIdx.x + (blockIdx.x * blockDim.x);
	int ty = threadIdx.y + (blockIdx.y * blockDim.y);
	int offset = tx + ty * blockDim.x*gridDim.x;
	int sh_offset = threadIdx.x + threadIdx.y * 16;

	sh_Tile[sh_offset * 4 + 0] = gpu_in[offset].x;
	sh_Tile[sh_offset * 4 + 1] = gpu_in[offset].y;
	sh_Tile[sh_offset * 4 + 2] = gpu_in[offset].z;
	sh_Tile[sh_offset * 4 + 3] = gpu_in[offset].w;
	
	__syncthreads();

	if(offset < 240*320)
	{	
		float color = 0.3 * (gpu_in[offset].x) + 0.6 * (gpu_in[offset].y) + 0.1 * (gpu_in[offset].z);
		gpu_in[offset].x = color;
		gpu_in[offset].y = color;
		gpu_in[offset].z = color;
		gpu_in[offset].w = 0;
	
		sh_Tile[sh_offset * 4 + 0] = (int) (0.3 * sh_Tile[sh_offset * 4 + 0] + 0.6 * sh_Tile[sh_offset * 4 + 1] + 0.1 * sh_Tile[sh_offset * 4 + 2]); 
	}
	__syncthreads();
	
	gpu_in[offset].x = sh_Tile[sh_offset * 4 + 0];
	gpu_in[offset].y = sh_Tile[sh_offset * 4 + 0];
	gpu_in[offset].z = sh_Tile[sh_offset * 4 + 0];
	gpu_in[offset].w = 0;
	*/

	
	
}
///////////////// CUDA function call wrapper /////////////////
float tograyscale(unsigned char *in, unsigned char * in_1)
{
	//uchar4 *gpu_in;

	unsigned char *iin_1;
	hipMalloc((void **)&iin_1, (240*320*sizeof(unsigned char)));
	
	hipArray *src;
    hipChannelFormatDesc floatTex = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
	//hipChannelFormatDesc floatTex = hipCreateChannelDesc<unsigned char>();
    hipMallocArray(&src, &floatTex, 240, 320);
    hipMemcpyToArray(src, 0, 0, in, 240 * 320 * 4, hipMemcpyHostToDevice);
    hipBindTextureToArray(texSrc, src, floatTex);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	////////////////////////// Time consuming Task //////////////////////////////////	
	//hipMalloc((void **)&gpu_in, (240*320*4*sizeof(unsigned char)));	
	//hipMemcpy(gpu_in, in, (240*320*4*sizeof(unsigned char)), hipMemcpyHostToDevice);

	

	dim3 grid(15,22);
	dim3 block(16,15);
	convert<<<grid,block>>>(iin_1);

	//hipMemcpy( in, gpu_in, (240*320*4*sizeof(unsigned char)), hipMemcpyDeviceToHost);
	/////////////////////////////////////////////////////////////////////////////////

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipMemcpy( in_1, iin_1, (240*320*sizeof(unsigned char)), hipMemcpyDeviceToHost);
	hipUnbindTexture(texSrc);
	
	return elapsedtime;	
}


