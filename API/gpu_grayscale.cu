#include "hip/hip_runtime.h"
#ifndef _CUDA_H
#define _CUDA_H
#include "hip/hip_runtime.h"
#endif

//#ifndef _API_H
//#define _API_H
#include "api.h"
//#endif

/////////////// Grayscale Cuda Fucntion ////////////////////
__global__ void convert(int width, int height, uchar4 *gpu_in)
{
	
	int tx = threadIdx.x + (blockIdx.x * blockDim.x);
	int ty = threadIdx.y + (blockIdx.y * blockDim.y);
	int offset = tx + ty * blockDim.x*gridDim.x;

	if(offset < width * height)
	{	
		float color = 0.3 * (gpu_in[offset].x) + 0.6 * (gpu_in[offset].y) + 0.1 * (gpu_in[offset].z);
		gpu_in[offset].x = color;
		gpu_in[offset].y = color;
		gpu_in[offset].z = color;
		gpu_in[offset].w = 0;
	}	
	
}
///////////////// CUDA function call wrapper /////////////////
gpu_error gpu_grayscale(int width, int height, unsigned char *in)
{
	uchar4 *gpu_in;
	float elapsedtime;
	hipEvent_t start, stop;
	gpu_error error = No_error;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	////////////////////////// Time consuming Task //////////////////////////////////	
	hipMalloc((void **)&gpu_in, (width * height * 4 * sizeof(unsigned char)));
	hipMemcpy(gpu_in, in, (width * height * 4 * sizeof(unsigned char)), hipMemcpyHostToDevice);
	error = checkCudaError();

	dim3 grid(18,18);
	dim3 block(16,16);
	convert<<<grid,block>>>(width, height, gpu_in);

	error = checkCudaError();
	hipMemcpy( in, gpu_in, (width * height * 4 * sizeof(unsigned char)), hipMemcpyDeviceToHost);
	/////////////////////////////////////////////////////////////////////////////////

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	hipFree(gpu_in);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return error;
	
}

