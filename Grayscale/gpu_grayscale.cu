#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
//#ifndef _API_H
//#define _API_H
#include "../api.h"
//#endif

/////////////// Grayscale Cuda Fucntion ////////////////////
__global__ void convert(int width, int height, unsigned char *gpu_in_1, unsigned char *gpu_in_4)
{
	int tx = threadIdx.x + (blockIdx.x * blockDim.x);
	int ty = threadIdx.y + (blockIdx.y * blockDim.y);
	int offset = tx + ty * blockDim.x*gridDim.x;
	int th_value = 20;

	if(offset < width * height)
	{
		float color = 0.3 * (gpu_in_4[offset * 4 + 0]) + 0.6 * (gpu_in_4[offset * 4 + 1]) + 0.1 * (gpu_in_4[offset * 4 + 2]);
		gpu_in_4[offset * 4 + 0] = color;
		gpu_in_4[offset * 4 + 1] = color;
		gpu_in_4[offset * 4 + 2] = color;
		gpu_in_4[offset * 4 + 3] = 0;
		//buffer[offset] = color;			// Dont know if it will work ---> It cant be done, It doesnt work
		if(color < th_value)		
			gpu_in_1[offset] = 0;			// There is really no need to call this function again for threshold when
		else						// all we have to do is copy from ouput->buffer to gpu->buffer and threshold.
			gpu_in_1[offset] = 255;			// I will be calcilating threshold here only and strothe result in gpu_in_1.

	}

}

///////////////// CUDA function call wrapper /////////////////
gpu_error_t gpu_grayscale(gpu_context_t *ctx)
{
	float elapsedtime;
	hipEvent_t start, stop;
	gpu_error_t error = GPU_OK;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	////////////////////////// Time consuming Task //////////////////////////////////	

	dim3 grid(18,18);
	dim3 block(16,16);
	convert<<<grid,block>>>( ctx->width, ctx->height, ctx->gpu_buffer_1, ctx->gpu_buffer_4);

	hipMemcpy( ctx->output_buffer_1, ctx->gpu_buffer_1, (ctx->width * ctx->height * sizeof(unsigned char)), hipMemcpyDeviceToHost);

	/////////////////////////////////////////////////////////////////////////////////

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return error;
}

